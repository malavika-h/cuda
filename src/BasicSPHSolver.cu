#include "hip/hip_runtime.h"
#include<stdio.h>
#include <vector>
#include <memory>
#include <hip/hip_runtime.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <hip/hip_vector_types.h>
#include "CUDAFunctions.cuh"
#include "DArray.h"
#include "Particles.h"
#include "SPHParticles.h"
#include "BaseSolver.h"
#include "BasicSPHSolver.h"

__device__ void contributeFluidDensity(float* density, const int i, float3* pos, float* mass, const int cellStart, const int cellEnd, const float radius)
{
	auto j = cellStart;
	while (j < cellEnd)
	{
		*density += mass[j] * cubic_spline_kernel(length(pos[i] - pos[j]), radius);
		++j;
	}
	return;
}

__device__ void contributeBoundaryDensity(float* density, const float3 pos_i, float3* pos, float* mass, const int cellStart, const int cellEnd, const float radius)
{
	auto j = cellStart;
	while (j < cellEnd)
	{
		*density += mass[j] * cubic_spline_kernel(length(pos_i - pos[j]), radius);
		++j;
	}
	return;
}

__global__ void computeDensity_CUDA(float* density, const int num,
	float3* posFluid, float* massFluid, int* cellStartFluid, 
	float3* posBoundary, float* massBoundary, int* cellStartBoundary,
	const int3 cellSize, const float cellLength, const float radius)
{
	const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	__syncthreads();
#pragma unroll
	for (auto m = 0; m < 27; __syncthreads(), ++m)
	{
		const auto cellID = particlePos2cellIdx(
			make_int3(posFluid[i] / cellLength) + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1), cellSize);
		if (cellID == (cellSize.x * cellSize.y * cellSize.z)) continue;
		contributeFluidDensity(&density[i], i, posFluid, massFluid, cellStartFluid[cellID], cellStartFluid[cellID + 1], radius);
		contributeBoundaryDensity(&density[i], posFluid[i], posBoundary, massBoundary, cellStartBoundary[cellID], cellStartBoundary[cellID + 1], radius);
	}
	return;
}

void BasicSPHSolver::computeDensity(std::shared_ptr<SPHParticles>& fluids, const std::shared_ptr<SPHParticles>& boundaries,
		const DArray<int>& cellStartFluid, const DArray<int>& cellStartBoundary, int3 cellSize, float cellLength, float radius) const
{
	int num = fluids->size();
	thrust::fill(thrust::device, fluids->getDensityPtr(), fluids->getDensityPtr() + num, 0);
	computeDensity_CUDA <<<(num - 1) / block_size + 1, block_size >>> (fluids->getDensityPtr(), num,
		fluids->getPosPtr(), fluids->getMassPtr(), cellStartFluid.addr(),
		boundaries->getPosPtr(), boundaries->getMassPtr(), cellStartBoundary.addr(),
		cellSize, cellLength, radius);
}

__global__ void enforceBoundary_CUDA(float3* pos, float3* vel, const int num, const float3 spaceSize)
{
	const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	if (pos[i].x <= spaceSize.x * .00f) { pos[i].x = spaceSize.x * .00f;	vel[i].x = fmaxf(vel[i].x, 0.0f); }
	if (pos[i].x >= spaceSize.x * .99f) { pos[i].x = spaceSize.x * .99f;	vel[i].x = fminf(vel[i].x, 0.0f); }
	if (pos[i].y <= spaceSize.y * .00f) { pos[i].y = spaceSize.y * .00f;	vel[i].y = fmaxf(vel[i].y, 0.0f); }
	if (pos[i].y >= spaceSize.y * .99f) { pos[i].y = spaceSize.y * .99f;	vel[i].y = fminf(vel[i].y, 0.0f); }
	if (pos[i].z <= spaceSize.z * .00f) { pos[i].z = spaceSize.z * .00f;	vel[i].z = fmaxf(vel[i].z, 0.0f); }
	if (pos[i].z >= spaceSize.z * .99f) { pos[i].z = spaceSize.z * .99f;	vel[i].z = fminf(vel[i].z, 0.0f); }
	return;
}

void BasicSPHSolver::advect(std::shared_ptr<SPHParticles>& fluids, float dt, float3 spaceSize) {
	fluids->advect(dt);
	enforceBoundary_CUDA <<<((fluids->size())-1)/block_size+1, block_size >>> (fluids->getPosPtr(), fluids->getVelPtr(), fluids->size(), spaceSize);
}

__global__ void computePressure_CUDA(float* pressure, float* density, const int num, const float rho0, const float stiff)
{
	const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	pressure[i] = stiff * (powf((density[i] / rho0), 7) - 1.0f);
	//clamp
	if (pressure[i] < 0.0f) pressure[i] = 0.0f;
	return;
}

__device__ void contributeFluidPressure(float3* a, const int i, float3* pos, float* mass, 
	float* density, float* pressure, const int cellStart, const int cellEnd, const float radius)
{
	auto j = cellStart;
	while (j < cellEnd)
	{
		if (i != j)
			* a += -mass[j] * 
			(pressure[i] / fmaxf(EPSILON, density[i] * density[i]) + pressure[j] / fmaxf(EPSILON, density[j] * density[j]))
			* cubic_spline_kernel_gradient(pos[i] - pos[j], radius);
		++j;
	}
	return;
}

__device__ void contributeBoundaryPressure(float3* a, const float3 pos_i, float3* pos, float* mass,
                                           const float density, const float pressure, const int cellStart, const int cellEnd, const float radius)
{
	auto j = cellStart;
	while (j < cellEnd)
	{
		*a += -mass[j] * (pressure / fmaxf(EPSILON, density * density)) * cubic_spline_kernel_gradient(pos_i - pos[j], radius);
		++j;
	}
	return;
}

__global__ void pressureForce_CUDA(float3* velFluid, float3* posFluid, float* massFluid, 
	float* density, float* pressure, const int num, int* cellStartFluid, 
	float3* posBoundary, float* massBoundary, int* cellStartBoundary,
	const int3 cellSize, const float cellLength, const float radius, const float dt)
{
	const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	auto a = make_float3(0.0f);
	__syncthreads();
#pragma unroll
	for (auto m = 0; m < 27; __syncthreads(), ++m)
	{
		const auto cellID = particlePos2cellIdx(
			make_int3(posFluid[i] / cellLength) + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1), cellSize);
		if (cellID == (cellSize.x * cellSize.y * cellSize.z)) continue;
		contributeFluidPressure(&a, i, posFluid, massFluid, density, pressure, cellStartFluid[cellID], cellStartFluid[cellID + 1], radius);
		contributeBoundaryPressure(&a, posFluid[i], posBoundary, massBoundary, density[i], pressure[i], cellStartBoundary[cellID], cellStartBoundary[cellID + 1], radius);
	}

	// dirty trick to prevent blowups in large dt
	if (length(a) > MAX_A)
		a = normalize(a) * MAX_A;

	velFluid[i] += a * dt;
	return;
}

void BasicSPHSolver::project(std::shared_ptr<SPHParticles>& fluids, const std::shared_ptr<SPHParticles>& boundaries,
		const DArray<int>& cellStartFluid, const DArray<int>& cellStartBoundary, float rho0, float stiff,
		int3 cellSize, float cellLength, float radius, float dt)
{
	int num = fluids->size();
	// step 1:: calculate density
	computeDensity(fluids, boundaries, cellStartFluid, cellStartBoundary, cellSize, cellLength, radius);
	// step 2: calculate pressure from density
	computePressure_CUDA <<<(num - 1) / block_size + 1, block_size >>> (fluids->getPressurePtr(), fluids->getDensityPtr(), num, rho0, stiff);
	// step 3: apply pressure force according to pressure
	pressureForce_CUDA <<<(num - 1) / block_size + 1, block_size >>> (fluids->getVelPtr(), fluids->getPosPtr(), fluids->getMassPtr(),
		fluids->getDensityPtr(), fluids->getPressurePtr(), num, cellStartFluid.addr(), 
		boundaries->getPosPtr(), boundaries->getMassPtr(), cellStartBoundary.addr(), 
		cellSize, cellLength, radius, dt);
}

__device__ void contributeViscosity(float3* a, const int i, float3* pos, float3* vel,
                                    float* mass, int j/*cellStart*/, const int cellEnd, const float rho0, const float radius) {
	while (j < cellEnd) {
		*a += mass[j] * ((vel[j] - vel[i]) / rho0) * viscosity_kernel_laplacian(length(pos[i] - pos[j]), radius);
		++j;
	}
	return;
}

__global__ void viscosity_CUDA(float3* deltaV, float3* vel, float3* pos,
                               float* mass, const int num, int* cellStart, const int3 cellSize,
                               const float cellLength, const float rho0, const float radius, const float visc, const float dt) {
	const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	auto a = make_float3(0.0f);
	__syncthreads();
#pragma unroll
	for (auto m = 0; m < 27; __syncthreads(), ++m) {
		const auto cellID = particlePos2cellIdx(make_int3(pos[i] / cellLength) + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1),
		                                 cellSize);
		if (cellID == (cellSize.x * cellSize.y * cellSize.z)) continue;
		contributeViscosity(&a, i, pos, vel, mass, cellStart[cellID], cellStart[cellID + 1], rho0, radius);
	}

	deltaV[i] = visc * a * dt;
	return;
}

void BasicSPHSolver::diffuse(std::shared_ptr<SPHParticles>& fluids, const DArray<int>& cellStartFluid,
		int3 cellSize, float cellLength, float rho0,
		float radius, float visc, float dt)
{
	int num = fluids->size();
	viscosity_CUDA <<<(num - 1) / block_size + 1, block_size >>> (bufferFloat3.addr(), fluids->getVelPtr(), fluids->getPosPtr(),
	                                                              fluids->getMassPtr(), num, cellStartFluid.addr(), cellSize, cellLength, 
	                                                              rho0, radius, visc, dt);
	thrust::transform(thrust::device,
		fluids->getVelPtr(), fluids->getVelPtr() + num,
		bufferFloat3.addr(),
		fluids->getVelPtr(),
		thrust::plus<float3>()
	);
}

void BasicSPHSolver::force(std::shared_ptr<SPHParticles>& fluids, float dt, float3 G)
{
	const auto dv = dt * G;
	thrust::transform(thrust::device,
		fluids->getVelPtr(), fluids->getVelPtr() + fluids->size(),
		fluids->getVelPtr(),
		ThrustHelper::plus<float3>(dv)
	);
}

void BasicSPHSolver::step(std::shared_ptr<SPHParticles>& fluids, const std::shared_ptr<SPHParticles>& boundaries,
		const DArray<int>& cellStartFluid, const DArray<int>& cellStartBoundary, float3 spaceSize,
		int3 cellSize, float cellLength, float radius, float dt,
		float rho0, float rhoB, float stiff, float visc, float3 G,
		float surfaceTensionIntensity, float airPressure)
{
	// step 1: non-pressure, non-viscosity force
	force(fluids, dt, G);
	// step 2: viscosity force, surface tension
	diffuse(fluids, cellStartFluid, cellSize,
		cellLength, rho0, radius,
		visc, dt);
	if (surfaceTensionIntensity > EPSILON || airPressure > EPSILON)
		handleSurface(fluids, boundaries,
			cellStartFluid, cellStartBoundary,
			rho0, rhoB, cellSize, cellLength, radius,
			dt, surfaceTensionIntensity, airPressure);
	// step 3: pressure force
	project(fluids, boundaries,
		cellStartFluid, cellStartBoundary, rho0, stiff,
		cellSize, cellLength, radius, dt);
	// step 4:: advection
	advect(fluids, dt, spaceSize);
}

void BasicSPHSolver::handleSurface(std::shared_ptr<SPHParticles>& fluids, const std::shared_ptr<SPHParticles>& boundaries,
		const DArray<int>& cellStartFluid, const DArray<int>& cellStartBoundary,
		float rho0, float rhoB, int3 cellSize, float cellLength, float radius,
		float dt, float surfaceTensionIntensity, float airPressure)
{
	// the free surface handling method is from
	// [2014][TOG][Robust Simulation of Small-Scale Thin Features in SPH-based Free Surface Flows]
	// use bufferFloat3 as color gradient array
	surfaceDetection(bufferFloat3, fluids, boundaries,
		cellStartFluid, cellStartBoundary,
		rho0, rhoB, cellSize, cellLength, radius);
	applySurfaceEffects(fluids, bufferFloat3, cellStartFluid, 
		rho0, cellSize, cellLength,	radius, dt, surfaceTensionIntensity, airPressure);
}

__device__ auto contributeColorGrad_fluid(float3& numerator, float& denominator, const int i, float3* pos, float* mass, int j, const int cellEnd, const float radius, const float rho0) -> void
{
	while (j < cellEnd)
	{
		numerator += mass[j] / rho0 * cubic_spline_kernel_gradient(pos[i] - pos[j], radius);
		denominator += mass[j] / rho0 * cubic_spline_kernel(length(pos[i] - pos[j]), radius);
		++j;
	}
	return;
}

__device__ void contributeColorGrad_boundary(float3& numerator, float& denominator, float3* pos_i, float3* pos, float* mass, int j, const int cellEnd, const float radius, const float rhoB)
{
	while (j < cellEnd)
	{
		numerator += mass[j] / rhoB * cubic_spline_kernel_gradient(*pos_i - pos[j], radius);
		denominator += mass[j] / rhoB * cubic_spline_kernel(length(*pos_i - pos[j]), radius);
		++j;
	}
	return;
}

__global__ void computeColorGrad_CUDA(float3* colorGrad, float3* posFluid, float* massFluid, const int num, int* cellStartFluid, const int3 cellSize,
                                      float3* posBoundary, float* massBoudnary, int* cellStartBoundary, const float cellLength, const float radius, const float rho0, const float rhoB)
{
	const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	auto c_g = make_float3(0.0f);
	auto denominator = 0.0f;
#pragma unroll
	for (auto m = 0; m < 27; __syncthreads(), ++m)
	{
		const auto cellID = particlePos2cellIdx(
			make_int3(posFluid[i] / cellLength) + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1), cellSize);
		if (cellID == (cellSize.x * cellSize.y * cellSize.z)) continue;
		contributeColorGrad_fluid(c_g, denominator, i, posFluid, massFluid, cellStartFluid[cellID], cellStartFluid[cellID + 1], radius, rho0);
		contributeColorGrad_boundary(c_g, denominator, &posFluid[i], posBoundary, massBoudnary, cellStartBoundary[cellID], cellStartBoundary[cellID + 1], radius, rhoB);
	}

	colorGrad[i] = c_g / fmaxf(EPSILON, denominator);
	return;
}

void BasicSPHSolver::surfaceDetection(DArray<float3>& colorGrad, const std::shared_ptr<SPHParticles>& fluids, const std::shared_ptr<SPHParticles>& boundaries,
		const DArray<int>& cellStartFluid, const DArray<int>& cellStartBoundary,
		float rho0, float rhoB, int3 cellSize, float cellLength, float radius)
{
	computeColorGrad_CUDA <<<(fluids->size()-1)/block_size+1, block_size>>> (colorGrad.addr(),
	                                                                         fluids->getPosPtr(), fluids->getMassPtr(), fluids->size(), cellStartFluid.addr(), 
	                                                                         cellSize, boundaries->getPosPtr(), 
	                                                                         boundaries->getMassPtr(), cellStartBoundary.addr(), cellLength,
	                                                                         radius, rho0, rhoB);
	return;
}

__device__ void contributeSurfaceTensionAndAirPressure(float3& a, const int i, float3* pos, float* mass,
                                                       float3* color_grad, int j, const int cellEnd, const float radius,
                                                       const float rho0, const float color_energy_coefficient, const float airPressure)
{
	while (j < cellEnd)
	{
		// surface tension
		a += 0.25f * mass[j] / (rho0 * rho0) * color_energy_coefficient
			* (dot(color_grad[i], color_grad[i]) + dot(color_grad[j], color_grad[j]))
			* surface_tension_kernel_gradient(pos[i] - pos[j], radius);
		// air pressure
		a += airPressure * mass[j] / (rho0 * rho0)
			* cubic_spline_kernel_gradient(pos[i] - pos[j], radius)
			/*following terms disable inner particles*/
			* length(color_grad[i]) / fmaxf(EPSILON, length(color_grad[i]));
		++j;
	}
	return;
}

__global__ void surfaceTensionAndAirPressure_CUDA(float3* vel, float3* pos_fluid, float* mass_fluid,
                                                  float3* color_grad, const int num, int* cellStart, const int3 cellSize, const float cellLength, const float radius, const float dt,
                                                  const float rho0, const float color_energy_coefficient, const float airPressure)
{
	const unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (i >= num) return;
	auto a = make_float3(0.0f);
#pragma unroll
	for (auto m = 0; m < 27; __syncthreads(), ++m)
	{
		const auto cellID = particlePos2cellIdx(
			make_int3(pos_fluid[i] / cellLength) + make_int3(m / 9 - 1, (m % 9) / 3 - 1, m % 3 - 1), cellSize);
		if (cellID == (cellSize.x * cellSize.y * cellSize.z)) continue;
		contributeSurfaceTensionAndAirPressure(a, i, pos_fluid, mass_fluid, color_grad, cellStart[cellID], cellStart[cellID + 1], 
		                                       radius, rho0, color_energy_coefficient, airPressure);
	}
	vel[i] += a * dt;
	return;
}

void BasicSPHSolver::applySurfaceEffects(std::shared_ptr<SPHParticles>& fluids, const DArray<float3>& colorGrad,
		const DArray<int>& cellStartFluid, float rho0, int3 cellSize, float cellLength,
		float radius, float dt, float surfaceTensionIntensity, float airPressure)
{
	int num = fluids->size();
	surfaceTensionAndAirPressure_CUDA <<<(num - 1) / block_size + 1, block_size>>> (fluids->getVelPtr(),
	                                                                                fluids->getPosPtr(), fluids->getMassPtr(), colorGrad.addr(),
	                                                                                num, cellStartFluid.addr(), cellSize, cellLength, radius, dt, rho0,
	                                                                                surfaceTensionIntensity, airPressure);
}
